#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>

#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <execution>
#include <iostream>
#include <numeric>
#include <random>

#include "brt.cuh"
#include "morton.cuh"
#include "sync.hpp"

namespace bm = benchmark;

static void BM_compute_morton_v2_only(bm::State &state) {
  const auto num_elements = state.range(0);
  const auto num_threads = state.range(1);
  float3 *u_input;
  Code_t *u_output;
  hipMallocManaged(&u_input, num_elements * sizeof(float3));
  hipMallocManaged(&u_output, num_elements * sizeof(Code_t));

  for (auto _ : state) {
    cuda_event_timer raii{state, true};

    const auto num_blocks = (num_elements + num_threads - 1) / num_threads;
    convertMortonOnly_v2<<<num_blocks, num_threads>>>(
        u_input, u_output, num_elements, Morton(0.0f, 1024.0f));
  }

  BENCH_CUDA_TRY(hipFree(u_input));
  BENCH_CUDA_TRY(hipFree(u_output));
}

static void BM_radixsort_morton_only(bm::State &state) {
  const auto num_elements = state.range(0);
  Code_t *u_mortons;
  Code_t *u_sorted_mortons;
  hipMallocManaged(&u_mortons, num_elements * sizeof(Code_t));
  hipMallocManaged(&u_sorted_mortons, num_elements * sizeof(Code_t));
  hipcub::CachingDeviceAllocator g_allocator(true);

  for (auto _ : state) {
    cuda_event_timer raii{state};

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    BENCH_CUDA_TRY(hipcub::DeviceRadixSort::SortKeys(
        d_temp_storage, temp_storage_bytes, u_mortons, u_sorted_mortons,
        num_elements));

    BENCH_CUDA_TRY(hipMalloc(&d_temp_storage, temp_storage_bytes));

    BENCH_CUDA_TRY(hipcub::DeviceRadixSort::SortKeys(
        d_temp_storage, temp_storage_bytes, u_mortons, u_sorted_mortons,
        num_elements));
  }

  BENCH_CUDA_TRY(hipFree(u_mortons));
  BENCH_CUDA_TRY(hipFree(u_sorted_mortons));
}

static void BM_unique_morton(bm::State &state) {
  const auto num_elements = state.range(0);
  Code_t *u_input;
  Code_t *u_output;
  int *u_num_selected_out;
  hipMallocManaged(&u_input, num_elements * sizeof(Code_t));
  hipMallocManaged(&u_output, num_elements * sizeof(Code_t));
  hipMallocManaged(&u_num_selected_out, 1 * sizeof(int));
  hipcub::CachingDeviceAllocator g_allocator(true);

  for (auto _ : state) {
    cuda_event_timer raii{state};

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    BENCH_CUDA_TRY(hipcub::DeviceSelect::Unique(d_temp_storage, temp_storage_bytes,
                                             u_input, u_output,
                                             u_num_selected_out, num_elements));

    BENCH_CUDA_TRY(hipMalloc(&d_temp_storage, temp_storage_bytes));

    BENCH_CUDA_TRY(hipcub::DeviceSelect::Unique(d_temp_storage, temp_storage_bytes,
                                             u_input, u_output,
                                             u_num_selected_out, num_elements));
  }

  BENCH_CUDA_TRY(hipFree(u_input));
  BENCH_CUDA_TRY(hipFree(u_output));
}

class RadixTreeFixture : public bm::Fixture {
public:
  void SetUp(const bm::State &state) override {
    const auto num_elements = state.range(0);
    hipMallocManaged(&u_input, num_elements * sizeof(float3));
    hipMallocManaged(&u_mortons, num_elements * sizeof(Code_t));
    BENCH_CUDA_TRY(hipDeviceSynchronize());

    // init random inputs
    const auto min_coord = 0.0f;
    const auto range = 1024.0f;

    thread_local std::mt19937 gen(114514); // NOLINT(cert-msc51-cpp)
    static std::uniform_real_distribution<float> dis(min_coord, range);

    std::generate_n(std::execution::par, u_input, num_elements, [&]() {
      return float3{dis(gen), dis(gen), dis(gen)};
    });

    std::transform(std::execution::par, u_input, u_input + num_elements,
                   u_mortons, [&](const auto &pt) {
                     return PointToCode(pt.x, pt.y, pt.z, min_coord, range);
                   });
    std::sort(std::execution::par, u_mortons, u_mortons + num_elements);

    const auto last_unique_it =
        std::unique(std::execution::par, u_mortons, u_mortons + num_elements);

    num_unique_keys = std::distance(u_mortons, last_unique_it);

    const auto num_brt_nodes = num_unique_keys - 1;
    hipMallocManaged(&u_inner_nondes, num_brt_nodes * sizeof(brt::InnerNodes));

    // prepare for the later benchmark
    constexpr auto num_threads = 256;
    const auto num_blocks = (num_unique_keys + num_threads - 1) / num_threads;
    BuildRadixTreeKernel<<<num_blocks, num_threads>>>(u_mortons, u_inner_nondes,
                                                      num_unique_keys);
    BENCH_CUDA_TRY(hipDeviceSynchronize());
  }

  void TearDown(const bm::State &state) override {
    BENCH_CUDA_TRY(hipFree(u_input));
    BENCH_CUDA_TRY(hipFree(u_mortons));
    BENCH_CUDA_TRY(hipFree(u_inner_nondes));
  }

protected:
  float3 *u_input;
  Code_t *u_mortons;
  int num_unique_keys;
  brt::InnerNodes *u_inner_nondes;
};

BENCHMARK_DEFINE_F(RadixTreeFixture, FooTest)(bm::State &st) {
  const auto num_elements = st.range(0);
  const auto num_threads = st.range(1);

  brt::InnerNodes *inner_nondes;
  hipMallocManaged(&inner_nondes, num_elements * sizeof(brt::InnerNodes));

  for (auto _ : st) {
    cuda_event_timer raii{st, true};

    const auto num_blocks = (num_elements + num_threads - 1) / num_threads;
    BuildRadixTreeKernel<<<num_blocks, num_threads>>>(u_mortons, inner_nondes,
                                                      num_unique_keys);
    bm::DoNotOptimize(inner_nondes);
  }

  BENCH_CUDA_TRY(hipFree(inner_nondes));
}

BENCHMARK_REGISTER_F(RadixTreeFixture, FooTest)
    ->ArgsProduct({{10'000'000}, {32, 64, 128, 256, 512, 1024}})
    ->Unit(bm::kMillisecond);

BENCHMARK(BM_compute_morton_v2_only)
    ->ArgsProduct({{10'000'000}, {32, 64, 128, 256, 512, 1024}})
    ->UseManualTime()
    ->Unit(bm::kMillisecond);

BENCHMARK(BM_radixsort_morton_only)
    ->Args({1'000'000})
    ->UseManualTime()
    ->Unit(bm::kMillisecond);

BENCHMARK(BM_unique_morton)
    ->Args({1'000'000})
    ->UseManualTime()
    ->Unit(bm::kMillisecond);

BENCHMARK_MAIN();
