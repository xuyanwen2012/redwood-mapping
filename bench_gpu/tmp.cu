#include "hip/hip_runtime.h"
#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <random>

#include "brt.cuh"
#include "morton.cuh"
#include "oct.cuh"

#define HANDLE_ERROR(err) (HandleCudaError(err, __FILE__, __LINE__))

inline void HandleCudaError(const hipError_t err, const char *file,
                            const int line) {
  if (err != hipSuccess) {
    const auto _ = fprintf(stderr, "CUDA Error: %s in %s at line %d\n",
                           hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
__global__ void EmptyKernel() {}

void GpuWarmUp() {
  EmptyKernel<<<1, 1>>>();
  HANDLE_ERROR(hipDeviceSynchronize());
}

void PrintCudaDeviceInfo() {
  int device_count;
  HANDLE_ERROR(hipGetDeviceCount(&device_count));

  for (int device = 0; device < device_count; ++device) {
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device);

    std::cout << "Device " << device << ": " << device_prop.name << std::endl;
    std::cout << "Compute Capability (SM version): " << device_prop.major << "."
              << device_prop.minor << std::endl;
  }
}

template <typename T> T *AllocateManaged(const size_t num_elements) {
  T *ptr;
  HANDLE_ERROR(hipMallocManaged(&ptr, num_elements * sizeof(T)));
  return ptr;
}

#define DEFINE_SYNC_KERNEL_WRAPPER(kernel_name, function_name, num_threads)    \
  template <typename... Args>                                                  \
  void function_name(const size_t num_items, Args... args) {                   \
    const auto num_blocks = (num_items + num_threads - 1) / num_threads;       \
    kernel_name<<<num_blocks, num_threads>>>(num_items, args...);              \
    HANDLE_ERROR(hipDeviceSynchronize());                                     \
  }

// Use these to generate a wrapper function for a GPU kernel
DEFINE_SYNC_KERNEL_WRAPPER(convertMortonOnly_v2, TransformMortonSync, 256)
DEFINE_SYNC_KERNEL_WRAPPER(BuildRadixTreeKernel, BuildRadixTreeSync, 256)
DEFINE_SYNC_KERNEL_WRAPPER(CalculateEdgeCountKernel, CalculateEdgeCountSync,
                           256)

// ---------------------
//        Kernels
// ---------------------

int main() {
  constexpr auto num_elements = 10'000'000;
  //   constexpr auto num_elements = 1280 * 720;
  // constexpr auto num_threads = 256;

  PrintCudaDeviceInfo();

  const auto u_input = AllocateManaged<float3>(num_elements);
  const auto u_mortons = AllocateManaged<Code_t>(num_elements);
  const auto u_mortons_alt = AllocateManaged<Code_t>(num_elements);
  const auto u_num_selected_out = AllocateManaged<int>(1);
  const auto u_inner_nodes = AllocateManaged<brt::InnerNodes>(num_elements);
  const auto u_edge_count = AllocateManaged<int>(num_elements);
  const auto u_oc_offset = AllocateManaged<int>(num_elements);

  // init random inputs
  constexpr auto min_coord = 0.0f;
  constexpr auto max_coord = 1024.0f;
  constexpr auto range = max_coord - min_coord;
  constexpr auto morton_functor = Morton(min_coord, range);

  thread_local std::mt19937 gen(114514); // NOLINT(cert-msc51-cpp)
  static std::uniform_real_distribution<float> dis(min_coord, range);

  std::generate_n(u_input, num_elements,
                  [&] { return make_float3(dis(gen), dis(gen), dis(gen)); });

  GpuWarmUp();

  TransformMortonSync(num_elements, u_input, u_mortons, morton_functor);
  // FooWrapper(num_elements, u_input, u_mortons, morton_functor);

  // print 10
  std::cout << "mortons:" << std::endl;
  for (auto i = 0; i < 10; ++i) {
    std::cout << u_mortons[i] << std::endl;
  }

  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  size_t last_temp_storage_bytes = 0;

  // Sort by morton codes
  {
    HANDLE_ERROR(hipcub::DeviceRadixSort::SortKeys(d_temp_storage,
                                                temp_storage_bytes, u_mortons,
                                                u_mortons_alt, num_elements));

    std::cout << "temp_storage_bytes:\t" << temp_storage_bytes << std::endl;
    HANDLE_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));

    HANDLE_ERROR(hipcub::DeviceRadixSort::SortKeys(d_temp_storage,
                                                temp_storage_bytes, u_mortons,
                                                u_mortons_alt, num_elements));

    HANDLE_ERROR(hipDeviceSynchronize());
  }

  // Unique morton codes
  {
    hipcub::DeviceSelect::Unique(d_temp_storage, temp_storage_bytes, u_mortons_alt,
                              u_mortons, u_num_selected_out, num_elements);

    if (last_temp_storage_bytes < temp_storage_bytes) {
      HANDLE_ERROR(hipFree(d_temp_storage));
      HANDLE_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));
      std::cout << "temp_storage_bytes:\t" << temp_storage_bytes << std::endl;
      last_temp_storage_bytes = temp_storage_bytes;
    }

    hipcub::DeviceSelect::Unique(d_temp_storage, temp_storage_bytes, u_mortons_alt,
                              u_mortons, u_num_selected_out, num_elements);

    HANDLE_ERROR(hipDeviceSynchronize());
  }

  const auto num_unique = *u_num_selected_out;
  std::cout << "num_unique:\t" << num_unique << std::endl;

  // print 10
  std::cout << "sorted:" << std::endl;
  for (auto i = 0; i < 10; ++i) {
    std::cout << u_mortons_alt[i] << std::endl;
  }

  // Build Radix Tree
  const auto num_brt_nodes = num_unique - 1;

  BuildRadixTreeSync(num_brt_nodes, u_mortons_alt, u_inner_nodes);

  // Print out some brt nodes
  for (auto i = 0; i < 10; ++i) {
    std::cout << i << ":\t" << u_inner_nodes[i].left << ", "
              << u_inner_nodes[i].right << "\t(" << u_inner_nodes[i].delta_node
              << ")" << std::endl;
  }

  CalculateEdgeCountSync(num_brt_nodes, u_edge_count, u_inner_nodes);

  // Print out some edge counts
  std::cout << "edge_count:" << std::endl;
  for (auto i = 0; i < 10; ++i) {
    std::cout << i << ":\t" << u_edge_count[i] << std::endl;
  }

  hipFree(u_input);
  hipFree(u_mortons);
  hipFree(u_mortons_alt);
  hipFree(u_num_selected_out);
  hipFree(u_inner_nodes);
  hipFree(u_edge_count);
  hipFree(u_oc_offset);
  hipFree(d_temp_storage);
  return 0;
}
