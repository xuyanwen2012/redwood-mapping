#include "hip/hip_runtime.h"
#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <random>

#include "brt.cuh"
#include "morton.cuh"
#include "oct.cuh"

#define HANDLE_ERROR(err) (HandleCudaError(err, __FILE__, __LINE__))

inline void HandleCudaError(const hipError_t err, const char *file,
                            const int line) {
  if (err != hipSuccess) {
    const auto _ = fprintf(stderr, "CUDA Error: %s in %s at line %d\n",
                           hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
__global__ void EmptyKernel() {}

void GpuWarmUp() {
  EmptyKernel<<<1, 1>>>();
  HANDLE_ERROR(hipDeviceSynchronize());
}

void PrintCudaDeviceInfo() {
  int device_count;
  HANDLE_ERROR(hipGetDeviceCount(&device_count));

  for (int device = 0; device < device_count; ++device) {
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device);

    std::cout << "Device " << device << ": " << device_prop.name << std::endl;
    std::cout << "Compute Capability (SM version): " << device_prop.major << "."
              << device_prop.minor << std::endl;
  }
}

template <typename T> T *AllocateManaged(const size_t num_elements) {
  T *ptr;
  HANDLE_ERROR(hipMallocManaged(&ptr, num_elements * sizeof(T)));
  return ptr;
}

#define DEFINE_SYNC_KERNEL_WRAPPER(kernel_name, function_name, num_threads)    \
  template <typename... Args>                                                  \
  void function_name(const size_t num_items, Args... args) {                   \
    const auto num_blocks = (num_items + num_threads - 1) / num_threads;       \
    kernel_name<<<num_blocks, num_threads>>>(num_items, args...);              \
    HANDLE_ERROR(hipDeviceSynchronize());                                     \
  }

#define DEFINE_CUB_WRAPPER(kernel, wrapper_name)                               \
  template <typename... Args> void wrapper_name(Args... args) {                \
    void *d_temp_storage = nullptr;                                            \
    size_t temp_storage_bytes = 0;                                             \
    kernel(d_temp_storage, temp_storage_bytes, args...);                       \
    hipMalloc(&d_temp_storage, temp_storage_bytes);                           \
    kernel(d_temp_storage, temp_storage_bytes, args...);                       \
    HANDLE_ERROR(hipDeviceSynchronize());                                     \
  }

// ---------------------
//        Kernels
// ---------------------

// Use these to generate a wrapper function for a GPU kernel
DEFINE_SYNC_KERNEL_WRAPPER(convertMortonOnly_v2, TransformMortonSync, 256)
DEFINE_SYNC_KERNEL_WRAPPER(BuildRadixTreeKernel, BuildRadixTreeSync, 256)
DEFINE_SYNC_KERNEL_WRAPPER(CalculateEdgeCountKernel, EdgeCountSync, 256)

DEFINE_CUB_WRAPPER(hipcub::DeviceRadixSort::SortKeys, CubRadixSort);
DEFINE_CUB_WRAPPER(hipcub::DeviceSelect::Unique, CubUnique);
DEFINE_CUB_WRAPPER(hipcub::DeviceScan::InclusiveSum, CubPrefixSum);

int main() {

  PrintCudaDeviceInfo();
  // constexpr auto num_elements = 10'000'000;
  constexpr auto num_elements = 1280 * 720;

  const auto u_input = AllocateManaged<float3>(num_elements);
  const auto u_mortons = AllocateManaged<Code_t>(num_elements);
  const auto u_mortons_alt = AllocateManaged<Code_t>(num_elements);
  const auto u_num_selected_out = AllocateManaged<int>(1);
  const auto u_inner_nodes = AllocateManaged<brt::InnerNodes>(num_elements);
  const auto u_edge_count = AllocateManaged<int>(num_elements);
  const auto u_oc_offset = AllocateManaged<int>(num_elements);

  // init random inputs
  constexpr auto min_coord = 0.0f;
  constexpr auto max_coord = 1024.0f;
  constexpr auto range = max_coord - min_coord;
  constexpr auto morton_functor = Morton(min_coord, range);
  thread_local std::mt19937 gen(114514); // NOLINT(cert-msc51-cpp)
  static std::uniform_real_distribution dis(min_coord, range); // <float>
  std::generate_n(u_input, num_elements,
                  [&] { return make_float3(dis(gen), dis(gen), dis(gen)); });

  GpuWarmUp();

  TransformMortonSync(num_elements, u_input, u_mortons, morton_functor);
  CubRadixSort(u_mortons, u_mortons_alt, num_elements);
  CubUnique(u_mortons_alt, u_mortons, u_num_selected_out, num_elements);

  const auto num_unique = *u_num_selected_out;
  const auto num_brt_nodes = num_unique - 1;

  BuildRadixTreeSync(num_brt_nodes, u_mortons_alt, u_inner_nodes);
  EdgeCountSync(num_brt_nodes, u_edge_count, u_inner_nodes);
  u_edge_count[0] = 1; // Root node counts as 1

  CubPrefixSum(u_edge_count, u_oc_offset + 1, num_brt_nodes);
  u_oc_offset[0] = 0;

  const auto num_oc_nodes = u_oc_offset[num_brt_nodes];
  std::cout << "num_oc_nodes:\t" << num_oc_nodes << std::endl;

  // Print out some stats
  std::cout << "num_unique:\t" << num_unique << std::endl;
  std::cout << "sorted (unique):" << std::endl;
  for (auto i = 0; i < 10; ++i) {
    std::cout << i << ":\t" << u_mortons[i] << std::endl;
  }
  for (auto i = 0; i < 10; ++i) {
    std::cout << i << ":\t" << u_inner_nodes[i].left << ", "
              << u_inner_nodes[i].right << "\t(" << u_inner_nodes[i].delta_node
              << ")" << std::endl;
  }
  std::cout << "edge_count:" << std::endl;
  for (auto i = 0; i < 10; ++i) {
    std::cout << i << ":\t" << u_edge_count[i] << std::endl;
  }
  std::cout << "oc_offset:" << std::endl;
  for (auto i = 0; i < 10; ++i) {
    std::cout << i << ":\t" << u_oc_offset[i] << std::endl;
  }

  hipFree(u_input);
  hipFree(u_mortons);
  hipFree(u_mortons_alt);
  hipFree(u_num_selected_out);
  hipFree(u_inner_nodes);
  hipFree(u_edge_count);
  hipFree(u_oc_offset);
  return 0;
}
