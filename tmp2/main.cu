
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function
__global__ void helloCUDA() { printf("Hello, CUDA World!\n"); }

int main() {
  // Launch the CUDA kernel with one block containing one thread
  helloCUDA<<<1, 1>>>();

  // Wait for the GPU to finish
  hipDeviceSynchronize();

  // Check for any CUDA errors
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
    return 1;
  }

  return 0;
}
